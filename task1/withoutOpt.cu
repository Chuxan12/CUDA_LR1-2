#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <iostream>
#include <cmath>

#define BLOCK_SIZE 16

using namespace std;

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

// Ядро для базового умножения матриц
__global__ void matrixMulKernel(float* C, const float* A, const float* B, int M, int N, int P) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < P) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * P + col];
        }
        C[row * P + col] = sum;
    }
}

// Хост-функция для базового умножения
void matrixMul(float* h_C, const float* h_A, const float* h_B, int M, int N, int P) {
    float* d_A, * d_B, * d_C;
    size_t sizeA = M * N * sizeof(float);
    size_t sizeB = N * P * sizeof(float);
    size_t sizeC = M * P * sizeof(float);

    CHECK_CUDA(hipMalloc(&d_A, sizeA));
    CHECK_CUDA(hipMalloc(&d_B, sizeB));
    CHECK_CUDA(hipMalloc(&d_C, sizeC));

    CHECK_CUDA(hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice));

    dim3 blockSize(16, 16);
    dim3 gridSize((P + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);

    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    CHECK_CUDA(hipEventRecord(start));
    matrixMulKernel << <gridSize, blockSize >> > (d_C, d_A, d_B, M, N, P);
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    cout << "Time: " << milliseconds << " ms" << endl;

    CHECK_CUDA(hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
}

int main() {
    int M = 16384, N = 16384, P = 16384; // Размеры матриц
    size_t sizeA = M * N * sizeof(float);
    size_t sizeB = N * P * sizeof(float);
    size_t sizeC = M * P * sizeof(float);

    float* h_A = new float[M * N];
    float* h_B = new float[N * P];
    float* h_C = new float[M * P];

    // Инициализация случайными значениями
    for (int i = 0; i < M * N; ++i) h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    for (int i = 0; i < N * P; ++i) h_B[i] = static_cast<float>(rand()) / RAND_MAX;

    // Запуск базовой версии
    matrixMul(h_C, h_A, h_B, M, N, P);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}