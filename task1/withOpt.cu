#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <iostream>
#include <cmath>

using namespace std;

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

#define TILE_SIZE 16

// Ядро с использованием Shared Memory
__global__ void matrixMulOptKernel(float* C, const float* A, const float* B, int M, int N, int P) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    float sum = 0.0f;

    for (int k = 0; k < (N + TILE_SIZE - 1) / TILE_SIZE; ++k) {
        int A_col = k * TILE_SIZE + tx;
        int B_row = k * TILE_SIZE + ty;

        As[ty][tx] = (row < M && A_col < N) ? A[row * N + A_col] : 0.0f;
        Bs[ty][tx] = (B_row < N && col < P) ? B[B_row * P + col] : 0.0f;

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i) {
            sum += As[ty][i] * Bs[i][tx];
        }

        __syncthreads();
    }

    if (row < M && col < P) {
        C[row * P + col] = sum;
    }
}

// Хост-функция для оптимизированного умножения
void matrixMul(float* h_C, const float* h_A, const float* h_B, int M, int N, int P) {
    float* d_A, * d_B, * d_C;
    size_t sizeA = M * N * sizeof(float);
    size_t sizeB = N * P * sizeof(float);
    size_t sizeC = M * P * sizeof(float);

    CHECK_CUDA(hipMalloc(&d_A, sizeA));
    CHECK_CUDA(hipMalloc(&d_B, sizeB));
    CHECK_CUDA(hipMalloc(&d_C, sizeC));

    CHECK_CUDA(hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice));

    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((P + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);

    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    CHECK_CUDA(hipEventRecord(start));
    matrixMulOptKernel << <gridSize, blockSize >> > (d_C, d_A, d_B, M, N, P);
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    cout << "Time: " << milliseconds << " ms" << endl;

    CHECK_CUDA(hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
}

int main() {
    int M = 16384, N = 16384, P = 16384; // Размеры матриц
    size_t sizeA = M * N * sizeof(float);
    size_t sizeB = N * P * sizeof(float);
    size_t sizeC = M * P * sizeof(float);

    float* h_A = new float[M * N];
    float* h_B = new float[N * P];
    float* h_C = new float[M * P];

    // Инициализация случайными значениями
    for (int i = 0; i < M * N; ++i) h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    for (int i = 0; i < N * P; ++i) h_B[i] = static_cast<float>(rand()) / RAND_MAX;

    // Запуск оптимизированной версии
    matrixMul(h_C, h_A, h_B, M, N, P);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}